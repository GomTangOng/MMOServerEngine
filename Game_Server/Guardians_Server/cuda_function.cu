#include "hip/hip_runtime.h"
#include "cuda_function.cuh"

struct TEST
{
	int a, b, c;

	__global__ void Test()
	{
		cout << "dddd\n" << endl;
	}
};

extern "C" void PrintPropertyDevice()
{
	hipDeviceProp_t prop;

	int dev_cnt;
	
	HANDLE_ERROR(hipGetDeviceCount(&dev_cnt));

	for (int i = 0; i < dev_cnt; ++i)
	{
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		cout << " --- General Inforamtion for device " << i << " ---\n";
		cout << "Name : " << prop.name << endl;
		cout << "Compute capability : " << prop.major << "." << prop.minor << endl;
		cout << "Clock rate : " << prop.clockRate << endl;
		cout << "Device copy overlap : ";
		
		if (prop.deviceOverlap)
			cout << "Enabled\n";
		else
			cout << "Disabled\n";
		cout << "Kernel execution timeout : ";
		if (prop.kernelExecTimeoutEnabled)
			cout << "Enabled\n";
		else
			cout << "Disabled\n";

		cout << " --- Memory Information for device " << i << " --- \n";
		cout << "Total global mem : " << prop.totalGlobalMem << endl;
		cout << "Total constant mem : " << prop.totalConstMem << endl;
		cout << "Max mem pitch : " << prop.memPitch << endl;
		cout << "Texture Alignment : " << prop.textureAlignment << endl;
		cout << " --- MP Information for device " << i << " --- \n";
		cout << "Multiprocessor count : " << prop.multiProcessorCount << endl;
		cout << "Shared mem per block : " << prop.sharedMemPerBlock << endl;
		cout << "registers per block : " << prop.regsPerBlock << endl;
		cout << "Threads in warp : " << prop.warpSize << endl;
		cout << "Max threads per block " << prop.maxThreadsPerBlock << endl;
		cout << "Max thread dimensions : ( " << prop.maxThreadsDim[0] << " , "
											 << prop.maxThreadsDim[1] << " , " 
											 << prop.maxThreadsDim[2] << " )" << endl;
		cout << "Max grid dimensions : ( " << prop.maxGridSize[0] << " , "
										   << prop.maxGridSize[1] << " , "
										   << prop.maxGridSize[2] << " )" << endl;
	}
}

extern "C" void CudaInit()
{

}

//CUDA_OBJECT *dev_obj;
//int blocks;
//
//__global__ void SphereCollsion(CUDA_OBJECT *obj_list, int obj_cnt, int my_id, bool *result)
//{
//	int t_id = threadIdx.x + blockIdx.x * blockDim.x;
//	int stride = blockDim.x * gridDim.x;
//
//	while (t_id < obj_cnt)
//	{
//		if (obj_list[my_id].SphereCollsion(&obj_list[t_id]))
//		{
//			*result = true;
//		}
//		t_id += stride;
//	}
//}
//
//__global__ void AABBCollision()
//{
//
//}
//
//__global__ void OBBCollision()
//{
//
//}
//
//extern "C" void Init_Cuda()
//{
//	hipDeviceProp_t prop;
//	hipGetDeviceProperties(&prop, 0);
//	blocks = prop.multiProcessorCount;
//
//	hipMalloc((void**)&dev_obj, sizeof(CUDA_OBJECT) * MAX_CLIENT);
//	//hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice);
//
//	//hipMalloc((void**)&dev_histo, 256 * sizeof(long));
//	//hipMemset(dev_histo, 0, 256 * sizeof(int));
//}
//
//extern "C" void Excute_Cuda(CUDA_OBJECT *obj, int obj_cnt, int my_id, bool *result)
//{
//	bool *dev_result;
//	hipMalloc((void**)&dev_result, sizeof(bool));
//	hipMemcpy(dev_obj, obj, sizeof(CUDA_OBJECT) * MAX_CLIENT, hipMemcpyHostToDevice);
//
//	SphereCollsion << <blocks * 2, ThreadsPerBlock >> > (dev_obj, obj_cnt, my_id, dev_result);
//
//	hipMemcpy(result, dev_result, sizeof(bool), hipMemcpyDeviceToHost);
//
//	hipFree(dev_result);
//}
//
//extern "C" void Release_Cuda()
//{
//	hipFree(dev_obj);
//}

//extern "C" void Init_Cuda2(CUDA_OBJECT *obj_list, int obj_cnt, int my_id, bool *result)
//{
//	static int cnt = 0;
//	bool d_r;
//	CUDA_OBJECT *d_obj_list;
//
//	hipMalloc((void**)&d_r, sizeof(bool));
//	hipMemcpy((void**)&d_obj_list, &obj_list, sizeof(CUDA_OBJECT) * MAX_CLIENT, hipMemcpyHostToDevice);
//
//	hipDeviceProp_t prop;
//	hipGetDeviceProperties(&prop, 0);
//	int blocks = prop.multiProcessorCount;
//
//	//SphereCollsion << <blocks * 2, 32 >> > (d_obj_list, obj_cnt, my_id, d_r);
//
//	cout << "Complete : " << ++cnt << endl;
//}